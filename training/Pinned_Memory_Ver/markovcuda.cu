#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include "markov.h"

// global variable
hipEvent_t start, stop;

// host matrices
int* major_high;
int* major_low;
int* minor_high;
int* minor_low;
int* major_chord;
int* minor_chord;

// host buffer
note_info* majorHighBuff;
note_info* majorLowBuff;
note_info* minorHighBuff;
note_info* minorLowBuff;

// memory transfer to device
note_info* device_majorHighBuff;
note_info* device_majorLowBuff;
note_info* device_minorHighBuff;
note_info* device_minorLowBuff;

int* device_majorHighNotes;
int* device_majorLowNotes;
int* device_minorHighNotes;
int* device_minorLowNotes;
int* device_majorChords;
int* device_minorChords;

void matrix_alloc() {
    // Allocation of major & minor notes transfer matrices //
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipHostAlloc(&major_high, sizeof(int) * (NUM_NOTE * NUM_NOTE), hipHostMallocMapped);
    hipHostAlloc(&major_low, sizeof(int) * (NUM_NOTE * NUM_NOTE), hipHostMallocMapped);
    hipHostAlloc(&minor_high, sizeof(int) * (NUM_NOTE * NUM_NOTE), hipHostMallocMapped);
    hipHostAlloc(&minor_low, sizeof(int) * (NUM_NOTE * NUM_NOTE), hipHostMallocMapped);

    // Allocation of major & minor chords transfer matrices //
    hipHostAlloc(&major_chord, sizeof(int) * (NUM_CHORD * NUM_CHORD), hipHostMallocMapped);
    hipHostAlloc(&minor_chord, sizeof(int) * (NUM_CHORD * NUM_CHORD), hipHostMallocMapped);

    // Allocation of buffer //
    hipHostAlloc(&majorHighBuff, sizeof(note_info) * BUFFER_LEN, hipHostMallocMapped);
    hipHostAlloc(&majorLowBuff, sizeof(note_info) * BUFFER_LEN, hipHostMallocMapped);
    hipHostAlloc(&minorHighBuff, sizeof(note_info) * BUFFER_LEN, hipHostMallocMapped);
    hipHostAlloc(&minorLowBuff, sizeof(note_info) * BUFFER_LEN, hipHostMallocMapped);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time spent for host memory allocation: " << elapsedTime << " ms\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void free_matrix() {
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipHostFree(major_high);
    hipHostFree(major_low);
    hipHostFree(minor_high);
    hipHostFree(minor_low);
    hipHostFree(major_chord);
    hipHostFree(minor_chord);

    hipHostFree(majorHighBuff);
    hipHostFree(majorLowBuff);
    hipHostFree(minorHighBuff);
    hipHostFree(minorLowBuff);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time spent for host memory free: " << elapsedTime << " ms\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void cuda_malloc() {
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc(&device_majorHighBuff, sizeof(note_info) * BUFFER_LEN);
    hipMalloc(&device_majorLowBuff, sizeof(note_info) * BUFFER_LEN);
    hipMalloc(&device_minorHighBuff, sizeof(note_info) * BUFFER_LEN);
    hipMalloc(&device_minorLowBuff, sizeof(note_info) * BUFFER_LEN);

    hipMalloc(&device_majorHighNotes, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMemset(device_majorHighNotes, 0, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMalloc(&device_majorLowNotes, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMemset(device_majorLowNotes, 0, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMalloc(&device_minorHighNotes, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMemset(device_minorHighNotes, 0, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMalloc(&device_minorLowNotes, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMemset(device_minorLowNotes, 0, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMalloc(&device_majorChords, sizeof(int) * NUM_CHORD * NUM_CHORD);
    hipMemset(device_majorChords, 0, sizeof(int) * NUM_CHORD * NUM_CHORD);
    hipMalloc(&device_minorChords, sizeof(int) * NUM_CHORD * NUM_CHORD);
    hipMemset(device_minorChords, 0, sizeof(int) * NUM_CHORD * NUM_CHORD);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time spent for device memory allocation: " << elapsedTime << " ms\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void cuda_free() {
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipFree(device_majorHighBuff);
    hipFree(device_majorLowBuff);
    hipFree(device_minorHighBuff);
    hipFree(device_minorLowBuff);

    hipFree(device_majorHighNotes);
    hipFree(device_majorLowNotes);
    hipFree(device_minorHighNotes);
    hipFree(device_minorLowNotes);
    hipFree(device_majorChords);
    hipFree(device_minorChords);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time spent for device memory free: " << elapsedTime << " ms\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void cuda_to_host() {
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMemcpy(major_high, device_majorHighNotes, sizeof(int) * NUM_NOTE * NUM_NOTE, hipMemcpyDeviceToHost);
    hipMemcpy(major_low, device_majorLowNotes, sizeof(int) * NUM_NOTE * NUM_NOTE, hipMemcpyDeviceToHost);
    hipMemcpy(minor_high, device_minorHighNotes, sizeof(int) * NUM_NOTE * NUM_NOTE, hipMemcpyDeviceToHost);
    hipMemcpy(minor_low, device_minorLowNotes, sizeof(int) * NUM_NOTE * NUM_NOTE, hipMemcpyDeviceToHost);
    hipMemcpy(major_chord, device_majorChords, sizeof(int) * NUM_CHORD * NUM_CHORD, hipMemcpyDeviceToHost);
    hipMemcpy(minor_chord, device_minorChords, sizeof(int) * NUM_CHORD * NUM_CHORD, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time spent for copy memory back: " << elapsedTime << " ms\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

__device__ inline int cuda_getNoteIndex(int curr_tone, int curr_dur, int prev_tone_1, int prev_dur_1, int tune) {
    int col = curr_tone * NUM_DURATION + curr_dur ;

    // If previous tone is chord, get top note and find closest
    if (prev_tone_1 >= CHORD_BASE) {
        prev_tone_1 = (prev_tone_1 - CHORD_BASE) / 144; // Get top note
        if (curr_tone == NUM_TONE - 1) { // if curr_tone is Rest
            prev_tone_1 = prev_tone_1 + 12 * (2 * tune);
        } else {
            prev_tone_1 = curr_tone - (curr_tone % 12) + prev_tone_1;
        }
    }

    int row;
    row = prev_tone_1 * NUM_DURATION + prev_dur_1;

    return row * NUM_NOTE + col;
}

__device__ inline int cuda_getChordIndex(int curr_tone, int prev_tone) {
    if (prev_tone >= CHORD_BASE) {
        prev_tone = prev_tone - CHORD_BASE;
    } 
    else if (prev_tone == NUM_TONE - 1) {
        return -1;
    }
    else {
        prev_tone = (prev_tone % 12) + (prev_tone % 12) * 12 + (prev_tone % 12) * 144;
    }
    return prev_tone * NUM_CHORD + (curr_tone - CHORD_BASE);
}

__global__ void note_kernel(note_info* device_Buff, int* device_Mat, int use_len) {
    int start, end;
    int tx = threadIdx.x;
    start = tx * (use_len / NUM_THREADS) + 1;
    if (tx == NUM_THREADS - 1) {
        end = use_len;
    } else {
        end = (tx + 1) * (use_len / NUM_THREADS);
    }

    int index;
    int curr_tone, curr_dur, prev_tone, prev_dur, tune;
    for (int i = start; i < end; i++) {
        curr_tone = device_Buff[i].tone;
        curr_tone = device_Buff[i].dur;
        prev_tone = device_Buff[i - 1].tone;
        prev_dur = device_Buff[i - 1].dur;
        tune = device_Buff[i].tune;
        if (curr_tone < CHORD_BASE && prev_tone != -1) {
            index = cuda_getNoteIndex(curr_tone, curr_dur, prev_tone, prev_dur, tune);
            if (index != -1) {
                atomicAdd(&device_Mat[index], 1);
            }
        }
    }
}

__global__ void chord_kernel(note_info* device_Buff, int* device_Mat, int use_len) {
    int start, end;
    int tx = threadIdx.x;
    start = tx * (use_len / NUM_THREADS) + 1;
    if (tx == NUM_THREADS - 1) {
        end = use_len;
    } else {
        end = (tx + 1) * (use_len / NUM_THREADS);
    }

    int index;
    int curr_tone, prev_tone;
    for (int i = start; i < end; i++) {
        curr_tone = device_Buff[i].tone;
        prev_tone = device_Buff[i - 1].tone;
        if (curr_tone >= CHORD_BASE && prev_tone != -1) {
            index = cuda_getChordIndex(curr_tone, prev_tone);
            if (index != -1) {
                atomicAdd(&device_Mat[index], 1);
            }
        }
    }    
}

void buffer_copy(note_info* major_high_buff, note_info* major_low_buff, int major_high_len, int major_low_len,
                 note_info* minor_high_buff, note_info* minor_low_buff, int minor_high_len, int minor_low_len) {
        float elapsedTime;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        hipMemcpy(device_majorHighBuff, major_high_buff, sizeof(note_info) * major_high_len, hipMemcpyHostToDevice);
        hipMemcpy(device_majorLowBuff, major_low_buff, sizeof(note_info) * major_low_len, hipMemcpyHostToDevice);

        hipMemcpy(device_minorHighBuff, minor_high_buff, sizeof(note_info) * minor_high_len, hipMemcpyHostToDevice);
        hipMemcpy(device_minorLowBuff, minor_low_buff, sizeof(note_info) * minor_low_len, hipMemcpyHostToDevice);
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        std::cout << "Time spent for buffer copy: " << elapsedTime << " ms\n";
        hipEventDestroy(start);
        hipEventDestroy(stop);
}

void cuda_note_count(int minor_high_len, int minor_low_len, int major_high_len, int major_low_len) {

    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    note_kernel<<<1, NUM_THREADS>>>(device_majorHighBuff, device_majorHighNotes, major_high_len);
    note_kernel<<<1, NUM_THREADS>>>(device_majorLowBuff, device_majorLowNotes, major_low_len);
    chord_kernel<<<1, NUM_THREADS>>>(device_majorHighBuff, device_majorChords, major_high_len);
    chord_kernel<<<1, NUM_THREADS>>>(device_majorLowBuff, device_majorChords, major_low_len);

    note_kernel<<<1, NUM_THREADS>>>(device_minorHighBuff,device_minorHighNotes, minor_high_len);
    note_kernel<<<1, NUM_THREADS>>>(device_minorLowBuff, device_minorLowNotes, minor_low_len);
    chord_kernel<<<1, NUM_THREADS>>>(device_minorHighBuff, device_minorChords, minor_high_len);
    chord_kernel<<<1, NUM_THREADS>>>(device_minorLowBuff, device_minorChords, minor_low_len);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Time spent for matrix generation: " << elapsedTime << " ms\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);
}