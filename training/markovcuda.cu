#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include "markov.h"

int buffer_size = sizeof(int) * BUFFER_LEN;
hipStream_t majorHighStream;
hipStream_t majorLowStream;
hipStream_t minorHighStream;
hipStream_t minorLowStream;
hipStream_t majorChordStream;
hipStream_t minorChordStream;

// host pinned-memory
int* majorHighTone;
int* majorHighDur;
int* majorLowTone;
int* majorLowDur;
int* minorHighTone;
int* minorHighDur;
int* minorLowTone;
int* minorLowDur;

// memory transfer to device
int* device_majorHighTone;
int* device_majorHighDur;
int* device_majorLowTone;
int* device_majorLowDur;
int* device_minorHighTone;
int* device_minorHighDur;
int* device_minorLowTone;
int* device_minorLowDur;

int* device_majorHighNotes;
int* device_majorLowNotes;
int* device_minorHighNotes;
int* device_minorLowNotes;
int* device_majorChords;
int* device_minorChords;

void cuda_pinned_alloc() {
    hipHostAlloc(&majorHighTone, buffer_size, hipHostMallocMapped);
    hipHostAlloc(&majorHighDur, buffer_size, hipHostMallocMapped);
    hipHostAlloc(&majorLowTone, buffer_size, hipHostMallocMapped);
    hipHostAlloc(&majorLowDur, buffer_size, hipHostMallocMapped);
    hipHostAlloc(&minorHighTone, buffer_size, hipHostMallocMapped);
    hipHostAlloc(&minorHighDur, buffer_size, hipHostMallocMapped);
    hipHostAlloc(&minorLowTone, buffer_size, hipHostMallocMapped);
    hipHostAlloc(&minorLowDur, buffer_size, hipHostMallocMapped);
}

void cuda_stream_create() {
    hipStreamCreate(&majorHighStream);
    hipStreamCreate(&majorLowStream);
    hipStreamCreate(&minorHighStream);
    hipStreamCreate(&minorLowStream);
    hipStreamCreate(&majorChordStream);
    hipStreamCreate(&minorChordStream);
}

void cuda_stream_destroy() {
    hipStreamDestroy(majorHighStream);
    hipStreamDestroy(majorLowStream);
    hipStreamDestroy(minorHighStream);
    hipStreamDestroy(minorLowStream);
    hipStreamDestroy(majorChordStream);
    hipStreamDestroy(minorChordStream);
}

void cuda_malloc() {
    hipMalloc(&device_majorHighTone, buffer_size);
    hipMalloc(&device_majorHighDur, buffer_size);
    hipMalloc(&device_majorLowTone, buffer_size);
    hipMalloc(&device_majorLowDur, buffer_size);
    hipMalloc(&device_minorHighTone, buffer_size);
    hipMalloc(&device_minorHighDur, buffer_size);
    hipMalloc(&device_minorLowTone, buffer_size);
    hipMalloc(&device_minorLowDur, buffer_size);

    hipMalloc(&device_majorHighNotes, sizeof(int) * NUM_TONE * NUM_TONE);
    hipMemsetAsync(device_majorHighNotes, 0, sizeof(int) * NUM_TONE * NUM_TONE, majorHighStream);
    hipMalloc(&device_majorLowNotes, sizeof(int) * NUM_TONE * NUM_TONE);
    hipMemsetAsync(device_majorLowNotes, 0, sizeof(int) * NUM_TONE * NUM_TONE, majorLowStream);
    hipMalloc(&device_minorHighNotes, sizeof(int) * NUM_TONE * NUM_TONE);
    hipMemsetAsync(device_minorHighNotes, 0, sizeof(int) * NUM_TONE * NUM_TONE, minorHighStream);
    hipMalloc(&device_minorLowNotes, sizeof(int) * NUM_TONE * NUM_TONE);
    hipMemsetAsync(device_minorLowNotes, 0, sizeof(int) * NUM_TONE * NUM_TONE, minorLowStream);
    hipMalloc(&device_majorChords, sizeof(int) * NUM_CHORD * NUM_CHORD);
    hipMemsetAsync(device_majorChords, 0, sizeof(int) * NUM_TONE * NUM_TONE, majorChordStream);
    hipMalloc(&device_minorChords, sizeof(int) * NUM_CHORD * NUM_CHORD);
    hipMemsetAsync(device_minorChords, 0, sizeof(int) * NUM_TONE * NUM_TONE, minorChordStream);
}

void cuda_host_free() {
    hipHostFree(majorHighTone);
    hipHostFree(majorHighDur);
    hipHostFree(majorLowTone);
    hipHostFree(majorLowDur);
    hipHostFree(minorHighTone);
    hipHostFree(minorHighDur);
    hipHostFree(minorLowTone);
    hipHostFree(minorLowDur);
}

void cuda_free() {
    hipFree(device_majorHighTone);
    hipFree(device_majorHighDur);
    hipFree(device_majorLowTone);
    hipFree(device_majorLowDur);
    hipFree(device_minorHighTone);
    hipFree(device_minorHighDur);
    hipFree(device_minorLowTone);
    hipFree(device_minorLowDur);

    hipFree(device_majorHighNotes);
    hipFree(device_majorLowNotes);
    hipFree(device_minorHighNotes);
    hipFree(device_minorLowNotes);
    hipFree(device_majorChords);
    hipFree(device_minorChords);
}

__device__
inline int cuda_getChordIndex(int curr_tone, int curr_dur, int prev_tone_1, int prev_dur_1, int tune) {
    int col = curr_tone * NUM_DURATION + curr_dur ;

    // If previous tone is chord, get top note and find closest
    if (prev_tone_1 >= CHORD_BASE) {
        prev_tone_1 = (prev_tone_1 - CHORD_BASE) / 144; // Get top note
        if (curr_tone == NUM_TONE - 1) { // if curr_tone is Rest
            prev_tone_1 = prev_tone_1 + 12 * (2 * tune);
        } else {
            prev_tone_1 = curr_tone - (curr_tone % 12) + prev_tone_1;
        }
    }

    int row;
    row = prev_tone_1 * NUM_DURATION + prev_dur_1;

    return row * NUM_NOTE + col;
}

__device__
inline int cuda_getChordIndex(int curr_tone, int prev_tone) {
    if (prev_tone >= CHORD_BASE) {
        prev_tone = prev_tone - CHORD_BASE;
    } 
    else if (prev_tone == NUM_TONE - 1) {
        return -1;
    }
    else {
        prev_tone = (prev_tone % 12) + (prev_tone % 12) * 12 + (prev_tone % 12) * 144;
    }
    return prev_tone * NUM_CHORD + (curr_tone - CHORD_BASE);
}

__global__ void note_kernel(int* device_Tone, int* device_Dur, int* device_mat, int use_len) {
    
}

__global__ void chord_kernel(int* device_tone, int* device_mat, int use_len) {
    
}

void cuda_note_count(int low_len, int high_len, int is_major) {
    if (is_major == 0)
    {     
        hipMemcpyAsync(device_minorHighTone, minorHighTone, buffer_size, hipMemcpyHostToDevice, minorHighStream);
        hipMemcpyAsync(device_minorHighDur, minorHighDur, buffer_size, hipMemcpyHostToDevice, minorHighStream);
        note_kernel<<1, NUM_THREADS, 0, minorHighStream>>(device_minorHighTone, device_minorHighDur, device_minorHigh, high_len);

        hipMemcpyAsync(device_minorLowTone, minorLowTone, buffer_size, hipMemcpyHostToDevice, minorLowStream);
        hipMemcpyAsync(device_minorLowDur, minorLowDur, buffer_size, hipMemcpyHostToDevice, minorLowStream);
        note_kernel<<1, NUM_THREADS, 0, minorHLowStream>>(device_minorLowTone, device_minorLowDur, device_minorLow, low_len);

        hipMemcpyAsync(device_minorLowTone, minorLowTone, buffer_size, hipMemcpyHostToDevice, minorChordStream);
        hipMemcpyAsync(device_minorHighTone, minorHighTone, buffer_size, hipMemcpyHostToDevice, minorChordStream);
        chord_kernel<<1, NUM_THREADS, 0, minorChordStream>>(device_minorLowTone, device_minorChords, low_len);
        chord_kernel<<1, NUM_THREADS, 0, minorChordStream>>(device_minorHighTone, device_minorChords, high_len);
    }
    else if (is_major == 1)
    {
        hipMemcpyAsync(device_majorHighTone, majorHighTone, buffer_size, hipMemcpyHostToDevice, majorHighStream);
        hipMemcpyAsync(device_majorHighDur, majorHighDur, buffer_size, hipMemcpyHostToDevice, majorHighStream);
        note_kernel<<1, NUM_THREADS, 0, majorHighStream>>(device_majorHighTone, device_majorHighDur, device_majorHigh, high_len);

        hipMemcpyAsync(device_majorLowTone, majorLowTone, buffer_size, hipMemcpyHostToDevice, majorLowStream);
        hipMemcpyAsync(device_majorLowDur, majorLowDur, buffer_size, hipMemcpyHostToDevice, majorLowStream);
        note_kernel<<1, NUM_THREADS, 0, majorHLowStream>>(device_minorLowTone, device_minorLowDur, device_majorLow, low_len);

        hipMemcpyAsync(device_majorLowTone, majorLowTone, buffer_size, hipMemcpyHostToDevice, majorChordStream);
        hipMemcpyAsync(device_minorHighTone, minorHighTone, buffer_size, hipMemcpyHostToDevice, majorChordStream);
        chord_kernel<<1, NUM_THREADS, 0, majorChordStream>>(device_majorLowTone, device_majorChords, low_len);
        chord_kernel<<1, NUM_THREADS, 0, majorChordStream>>(device_majorHighTone, device_majorChords, high_len);
    }
}

void cuda_stream_synch(int is_major) {
    if (is_major == 0)
    {
        hipStreamSynchronize(minorHighStream);
        hipStreamSynchronize(minorLowStream);
        hipStreamSynchronize(minorChordStream);
    }
    else if (is_major == 1)
    {
        hipStreamSynchronize(majorHighStream);
        hipStreamSynchronize(majorLowStream);
        hipStreamSynchronize(majorChordStream);
    }
}   

void cuda_to_host() {
    hipMemcpyAsync(major_high, device_majorHighNotes, sizeof(int) * NUM_TONE * NUM_TONE, hipMemcpyDeviceToHost, majorHighStream);
    hipMemcpyAsync(major_low, device_majorLowNotes, sizeof(int) * NUM_TONE * NUM_TONE, hipMemcpyDeviceToHost, majorLowStream);
    hipMemcpyAsync(minor_high, device_minorHighNotes, sizeof(int) * NUM_TONE * NUM_TONE, hipMemcpyDeviceToHost, minorHighStream);
    hipMemcpyAsync(minor_low, device_minorLowNotes, sizeof(int) * NUM_TONE * NUM_TONE, hipMemcpyDeviceToHost, minorLowStream);
    hipMemcpyAsync(major_chord, device_majorChords, sizeof(int) * NUM_CHORD * NUM_CHORD, hipMemcpyDeviceToHost, majorChordStream);
    hipMemcpyAsync(minor_chord, device_minorChords, sizeof(int) * NUM_CHORD * NUM_CHORD, hipMemcpyDeviceToHost, minorChordStream);
}

bool cuda_matrix_generation(char* major_path, char* minor_path) {
    std::cout << "Start parsing major & minor txt files" << std::endl;
    std::ifstream major_file(major_path);
    if (!major_file) {
        std::cerr << "Cannot open " << major_path << " !" <<std::endl;
        return false;
    }
    std::ifstream minor_file(minor_path);
    if (!minor_file) {
        std::cerr << "Cannot open " << minor_path << " !" <<std::endl;
        return false;        
    }

    cuda_stream_create();
    cuda_pinned_alloc();
    cuda_malloc();

    int tune = 1;
    int curr_tone = -1;
    int curr_dur = -1;
    int prev_tone_1 = -1;
    int prev_dur_1 = -1;
    int is_major = 1;
    int newMidi_flag = 0;

    int high_len = 0;
    int low_len = 0;
    int num_finished = 0;
    int is_major = 1;

    size_t split_idx;
    int cell_idx;
    std::string line;

    while (num_finished != 2) {
        if (is_major == 1) // major file
        { 
            if (!std::getline(major_file, line)) {
                cuda_note_count(low_len, high_len, is_major);
                cuda_stream_synch(is_major);
                high_len = 0;
                low_len = 0;
                is_major = 0;
                num_finished++;
                continue;
            }
            if (line.find('S') != std::string::npos && newMidi_flag == 0) { // start of a midi file
                curr_tone = -1;
                curr_dur = -1; 
                prev_tone_1 = -1; 
                prev_dur_1 = -1; 
                newMidi_flag = 1;
                continue;
            }
            if (line.find('L') != std::string::npos) { // low melody
                tune = 1;
                continue;
            }
            else if (line.find('H') != std::string::npos) { // high melody
                tune = 2;
                continue;
            }
            else if (line.find('X') != std::string::npos) { // end of a midi file
                newMidi_flag = 0;
                continue;
            }
            else if ((split_idx = line.find(' ')) != std::string::npos) {
                curr_tone = std::stoi(line.substr(0, split_idx));
                curr_dur = std::stoi(line.substr(split_idx));                
                if (tune == 1) {
                    majorLowTone[low_len] = curr_tone;
                    majorLowDur[low_len] = curr_dur;
                    low_len++;
                } 
                else if (tune == 2) {
                    majorHighTone[high_len] = curr_tone;
                    majorHighDur[high_len] = curr_dur;
                    high_len++;
                }

                prev_tone_1 = curr_tone;
                prev_dur_1 = curr_dur;

                if (high_len > BUFFER_LEN || low_len > BUFFER_LEN) {
                    cuda_notec_ount(low_len, high_len, is_major);
                    cuda_stream_synch(is_major);
                    high_len = 0;
                    low_len = 0;
                    is_major = 0;
                    continue;
                }
            }
        }
        else // minor file
        {
            if (!std::getline(minor_file, line)) {
                cuda_note_count(low_len, high_len, is_major);
                cuda_stream_synch(is_major);
                high_len = 0;
                low_len = 0;
                is_major = 1;
                num_finished++;
                continue;
            }
            if (line.find('S') != std::string::npos && newMidi_flag == 0) { // start of a midi file
                curr_tone = -1;
                curr_dur = -1; 
                prev_tone_1 = -1; 
                prev_dur_1 = -1; 
                newMidi_flag = 1;
                continue;
            }
            if (line.find('L') != std::string::npos) { // low melody
                tune = 1;
                continue;
            }
            else if (line.find('H') != std::string::npos) { // high melody
                tune = 2;
                continue;
            }
            else if (line.find('X') != std::string::npos) { // end of a midi file
                newMidi_flag = 0;
                continue;
            }
            else if ((split_idx = line.find(' ')) != std::string::npos) {
                curr_tone = std::stoi(line.substr(0, split_idx));
                curr_dur = std::stoi(line.substr(split_idx));                
                if (tune == 1) {
                    minorLowTone[low_len] = curr_tone;
                    minorLowDur[low_len] = curr_dur;
                    low_len++;
                } 
                else if (tune == 2) {
                    minorHighTone[high_len] = curr_tone;
                    minorHighDur[high_len] = curr_dur;
                    high_len++;
                }

                prev_tone_1 = curr_tone;
                prev_dur_1 = curr_dur;

                if (high_len > BUFFER_LEN || low_len > BUFFER_LEN) {
                    cuda_note_count(low_len, high_len, is_major);
                    cuda_stream_synch(is_major);
                    high_len = 0;
                    low_len = 0;
                    is_major = 1;
                    continue;
                }
            }
        }
    }
    
    // copy memory back to host
    cuda_to_host();
    cuda_stream_synch(0);
    cuda_stream_synch(1);

    // free pinned_memory
    cuda_host_free();
    cuda_free();
    cuda_stream_destroy();
}