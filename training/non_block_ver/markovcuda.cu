#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include "markov.h"

hipStream_t majorHighStream;
hipStream_t majorLowStream;
hipStream_t minorHighStream;
hipStream_t minorLowStream;
hipStream_t majorHighChordStream;
hipStream_t majorLowChordStream;
hipStream_t minorHighChordStream;
hipStream_t minorLowChordStream;

// memory transfer to device
int* device_majorHighTone;
int* device_majorHighDur;
int* device_majorLowTone;
int* device_majorLowDur;
int* device_minorHighTone;
int* device_minorHighDur;
int* device_minorLowTone;
int* device_minorLowDur;

int* device_majorHighNotes;
int* device_majorLowNotes;
int* device_minorHighNotes;
int* device_minorLowNotes;
int* device_majorChords;
int* device_minorChords;

void cuda_stream_create() {
    hipStreamCreate(&majorHighStream);
    hipStreamCreate(&majorLowStream);
    hipStreamCreate(&minorHighStream);
    hipStreamCreate(&minorLowStream);
    hipStreamCreate(&majorHighChordStream);
    hipStreamCreate(&majorLowChordStream);
    hipStreamCreate(&minorHighChordStream);
    hipStreamCreate(&minorLowChordStream);
}

void cuda_stream_destroy() {
    hipStreamDestroy(majorHighStream);
    hipStreamDestroy(majorLowStream);
    hipStreamDestroy(minorHighStream);
    hipStreamDestroy(minorLowStream);
    hipStreamDestroy(majorHighChordStream);
    hipStreamDestroy(majorLowChordStream);
    hipStreamDestroy(minorHighChordStream);
    hipStreamDestroy(minorLowChordStream);
}

void cuda_malloc() {
    hipMalloc(&device_majorHighTone, sizeof(int) * BUFFER_LEN);
    hipMalloc(&device_majorHighDur, sizeof(int) * BUFFER_LEN);
    hipMalloc(&device_majorLowTone, sizeof(int) * BUFFER_LEN);
    hipMalloc(&device_majorLowDur, sizeof(int) * BUFFER_LEN);
    hipMalloc(&device_minorHighTone, sizeof(int) * BUFFER_LEN);
    hipMalloc(&device_minorHighDur, sizeof(int) * BUFFER_LEN);
    hipMalloc(&device_minorLowTone, sizeof(int) * BUFFER_LEN);
    hipMalloc(&device_minorLowDur, sizeof(int) * BUFFER_LEN);

    hipMalloc(&device_majorHighNotes, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMemsetAsync(device_majorHighNotes, 0, sizeof(int) * NUM_NOTE * NUM_NOTE, majorHighStream);
    hipMalloc(&device_majorLowNotes, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMemsetAsync(device_majorLowNotes, 0, sizeof(int) * NUM_NOTE * NUM_NOTE, majorLowStream);
    hipMalloc(&device_minorHighNotes, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMemsetAsync(device_minorHighNotes, 0, sizeof(int) * NUM_NOTE * NUM_NOTE, minorHighStream);
    hipMalloc(&device_minorLowNotes, sizeof(int) * NUM_NOTE * NUM_NOTE);
    hipMemsetAsync(device_minorLowNotes, 0, sizeof(int) * NUM_NOTE * NUM_NOTE, minorLowStream);
    hipMalloc(&device_majorChords, sizeof(int) * NUM_CHORD * NUM_CHORD);
    hipMemsetAsync(device_majorChords, 0, sizeof(int) * NUM_CHORD * NUM_CHORD, majorHighChordStream);
    hipMalloc(&device_minorChords, sizeof(int) * NUM_CHORD * NUM_CHORD);
    hipMemsetAsync(device_minorChords, 0, sizeof(int) * NUM_CHORD * NUM_CHORD, minorHighChordStream);
}

void cuda_free() {
    hipFree(device_majorHighTone);
    hipFree(device_majorHighDur);
    hipFree(device_majorLowTone);
    hipFree(device_majorLowDur);
    hipFree(device_minorHighTone);
    hipFree(device_minorHighDur);
    hipFree(device_minorLowTone);
    hipFree(device_minorLowDur);

    hipFree(device_majorHighNotes);
    hipFree(device_majorLowNotes);
    hipFree(device_minorHighNotes);
    hipFree(device_minorLowNotes);
    hipFree(device_majorChords);
    hipFree(device_minorChords);
}

void cuda_stream_synch(int is_major) {
    if (is_major == 0)
    {
        hipStreamSynchronize(minorHighStream);
        hipStreamSynchronize(minorLowStream);
        hipStreamSynchronize(minorHighChordStream);
        hipStreamSynchronize(minorLowChordStream);
    }
    else if (is_major == 1)
    {
        hipStreamSynchronize(majorHighStream);
        hipStreamSynchronize(majorLowStream);
        hipStreamSynchronize(majorHighChordStream);
        hipStreamSynchronize(majorLowChordStream);
    }
}   

void cuda_to_host() {
    hipMemcpyAsync(major_high, device_majorHighNotes, sizeof(int) * NUM_NOTE * NUM_NOTE, hipMemcpyDeviceToHost, majorHighStream);
    hipMemcpyAsync(major_low, device_majorLowNotes, sizeof(int) * NUM_NOTE * NUM_NOTE, hipMemcpyDeviceToHost, majorLowStream);
    hipMemcpyAsync(minor_high, device_minorHighNotes, sizeof(int) * NUM_NOTE * NUM_NOTE, hipMemcpyDeviceToHost, minorHighStream);
    hipMemcpyAsync(minor_low, device_minorLowNotes, sizeof(int) * NUM_NOTE * NUM_NOTE, hipMemcpyDeviceToHost, minorLowStream);
    hipMemcpyAsync(major_chord, device_majorChords, sizeof(int) * NUM_CHORD * NUM_CHORD, hipMemcpyDeviceToHost, majorHighChordStream);
    hipMemcpyAsync(minor_chord, device_minorChords, sizeof(int) * NUM_CHORD * NUM_CHORD, hipMemcpyDeviceToHost, minorHighChordStream);
}

__device__ inline int cuda_getNoteIndex(int curr_tone, int curr_dur, int prev_tone_1, int prev_dur_1, int tune) {
    int col = curr_tone * NUM_DURATION + curr_dur ;

    // If previous tone is chord, get top note and find closest
    if (prev_tone_1 >= CHORD_BASE) {
        prev_tone_1 = (prev_tone_1 - CHORD_BASE) / 144; // Get top note
        if (curr_tone == NUM_TONE - 1) { // if curr_tone is Rest
            prev_tone_1 = prev_tone_1 + 12 * (2 * tune);
        } else {
            prev_tone_1 = curr_tone - (curr_tone % 12) + prev_tone_1;
        }
    }

    int row;
    row = prev_tone_1 * NUM_DURATION + prev_dur_1;

    return row * NUM_NOTE + col;
}

__device__ inline int cuda_getChordIndex(int curr_tone, int prev_tone) {
    if (prev_tone >= CHORD_BASE) {
        prev_tone = prev_tone - CHORD_BASE;
    } 
    else if (prev_tone == NUM_TONE - 1) {
        return -1;
    }
    else {
        prev_tone = (prev_tone % 12) + (prev_tone % 12) * 12 + (prev_tone % 12) * 144;
    }
    return prev_tone * NUM_CHORD + (curr_tone - CHORD_BASE);
}

__global__ void note_kernel(int* device_Tone, int* device_Dur, int* device_mat, int use_len, int tune) {
    int start, end;
    int tx = threadIdx.x;
    start = tx * (use_len / NUM_THREADS) + 1;
    if (tx == NUM_THREADS - 1) {
        end = use_len;
    } else {
        end = (tx + 1) * (use_len / NUM_THREADS);
    }

    int index;
    int curr_Tone, curr_Dur, prev_Tone, prev_Dur;
    for (int i = start; i < end; i++) {
        curr_Tone = device_Tone[i];
        curr_Dur = device_Dur[i];
        prev_Tone = device_Tone[i - 1];
        prev_Dur = device_Dur[i - 1];
        if (curr_Tone < CHORD_BASE && prev_Tone != -1) {
            index = cuda_getNoteIndex(curr_Tone, curr_Dur, prev_Tone, prev_Dur, tune);
            if (index != -1) {
                atomicAdd(&device_mat[index], 1);
            }
        }
    }
}

__global__ void chord_kernel(int* device_Tone, int* device_mat, int use_len) {
    int start, end;
    int tx = threadIdx.x;
    start = tx * (use_len / NUM_THREADS) + 1;
    if (tx == NUM_THREADS - 1) {
        end = use_len;
    } else {
        end = (tx + 1) * (use_len / NUM_THREADS);
    }

    int index;
    int curr_Tone, prev_Tone;
    for (int i = start; i < end; i++) {
        curr_Tone = device_Tone[i];
        prev_Tone = device_Tone[i - 1];
        if (curr_Tone >= CHORD_BASE && prev_Tone != -1) {
            index = cuda_getChordIndex(curr_Tone, prev_Tone);
            if (index != -1) {
                atomicAdd(&device_mat[index], 1);
            }
        }
    }    
}

void cuda_note_count(int* high_tone, int* hign_dur, int* low_tone, int* low_dur, int high_len, int low_len, int is_major, int tune) {
    if (is_major == 0)
    {     
        hipMemcpyAsync(device_minorHighTone, high_tone, sizeof(int) * high_len, hipMemcpyHostToDevice, minorHighStream);
        hipMemcpyAsync(device_minorHighDur, hign_dur, sizeof(int) * high_len, hipMemcpyHostToDevice, minorHighStream);
        note_kernel<<<1, NUM_THREADS, 0, minorHighStream>>>(device_minorHighTone, device_minorHighDur, device_minorHighNotes, high_len, tune);

        hipMemcpyAsync(device_minorLowTone, low_tone, sizeof(int) * low_len, hipMemcpyHostToDevice, minorLowStream);
        hipMemcpyAsync(device_minorLowDur, low_dur, sizeof(int) * low_len, hipMemcpyHostToDevice, minorLowStream);
        note_kernel<<<1, NUM_THREADS, 0, minorLowStream>>>(device_minorLowTone, device_minorLowDur, device_minorLowNotes, low_len, tune);

        hipMemcpyAsync(device_minorLowTone, low_tone, sizeof(int) * low_len, hipMemcpyHostToDevice, minorLowChordStream);
        hipMemcpyAsync(device_minorHighTone, high_tone, sizeof(int) * high_len, hipMemcpyHostToDevice, minorHighChordStream);
        chord_kernel<<<1, NUM_THREADS, 0, minorLowChordStream>>>(device_minorLowTone, device_minorChords, low_len);
        chord_kernel<<<1, NUM_THREADS, 0, minorHighChordStream>>>(device_minorHighTone, device_minorChords, high_len);
    }
    else if (is_major == 1)
    {
        hipMemcpyAsync(device_majorHighTone, high_tone, sizeof(int) * high_len, hipMemcpyHostToDevice, majorHighStream);
        hipMemcpyAsync(device_majorHighDur, hign_dur, sizeof(int) * high_len, hipMemcpyHostToDevice, majorHighStream);
        note_kernel<<<1, NUM_THREADS, 0, majorHighStream>>>(device_majorHighTone, device_majorHighDur, device_majorHighNotes, high_len, tune);

        hipMemcpyAsync(device_majorLowTone, low_tone, sizeof(int) * low_len, hipMemcpyHostToDevice, majorLowStream);
        hipMemcpyAsync(device_majorLowDur, low_dur, sizeof(int) * low_len, hipMemcpyHostToDevice, majorLowStream);
        note_kernel<<<1, NUM_THREADS, 0, majorLowStream>>>(device_minorLowTone, device_minorLowDur, device_majorLowNotes, low_len, tune);

        hipMemcpyAsync(device_majorLowTone, low_tone, sizeof(int) * low_len, hipMemcpyHostToDevice, majorHighChordStream);
        hipMemcpyAsync(device_minorHighTone, high_tone, sizeof(int) * high_len, hipMemcpyHostToDevice, majorLowChordStream);
        chord_kernel<<<1, NUM_THREADS, 0, majorLowChordStream>>>(device_majorLowTone, device_majorChords, low_len);
        chord_kernel<<<1, NUM_THREADS, 0, majorHighChordStream>>>(device_majorHighTone, device_majorChords, high_len);
    }
}